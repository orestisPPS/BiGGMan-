
//
// Created by hal9000 on 7/26/23.
//

#include "MemoryManagementCUDA.cuh"

namespace LinearAlgebraCUDA {

    void MemoryManagementCUDA::allocateDeviceMemory(double** d_array, int size) {
        hipError_t err = hipMalloc((void**)d_array, size * sizeof(double));
        if (err != hipSuccess) {
            throw std::runtime_error("Error allocating memory: " + std::string(hipGetErrorString(err)));
        }
    }

    void MemoryManagementCUDA::copyToDevice(double* d_array, const double* h_array, int size) {
        hipError_t err = hipMemcpy(d_array, h_array, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            throw std::runtime_error("Error copying to device: " + std::string(hipGetErrorString(err)));
        }
    }

    void MemoryManagementCUDA::copyToHost(double* h_array, const double* d_array, int size) {
        hipError_t err = hipMemcpy(h_array, d_array, size * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            throw std::runtime_error("Error copying to host: " + std::string(hipGetErrorString(err)));
        }
    }

    void MemoryManagementCUDA::freeDeviceMemory(double* d_array) {
        hipError_t err = hipFree(d_array);
        if (err != hipSuccess) {
            throw std::runtime_error("Error deallocating memory: " + std::string(hipGetErrorString(err)));
        }
    }
    
}
